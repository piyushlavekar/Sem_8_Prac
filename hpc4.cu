#include <iostream>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void add(int* A, int* B, int* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 4;
    int A[N], B[N], C[N];

    for (int i = 0; i < N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    int *dA, *dB, *dC;
    size_t size = N * sizeof(int);
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);

    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    add<<<1, N>>>(dA, dB, dC, N);
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    cout << "A: "; for (int i : A) cout << i << " ";
    cout << "\nB: "; for (int i : B) cout << i << " ";
    cout << "\nC: "; for (int i : C) cout << i << " ";

    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;
}


// A = (int*)malloc(size);
// B = (int*)malloc(size);
// C = (int*)malloc(size);
